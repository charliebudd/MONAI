#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include "utils/tensor_description.h"

#include <iostream>

#define BLOCK_SIZE 32
#define CAPACITY_BITS 4
#define CAPACITY_MASK 0xF
#define HEIGHT_MAX ((1 << CAPACITY_BITS) - 1)

__constant__ int c_width;
__constant__ int c_height;
__constant__ int c_element_count;
__constant__ int c_block_count_x;

__device__ inline int get_edge(int edges, int index)
{
    return (edges >> (index * CAPACITY_BITS)) & CAPACITY_MASK;
}

__device__ inline int set_edge(int edges, int index, int edge)
{
    return edges |= (edge & CAPACITY_MASK) << index * CAPACITY_BITS;
}

//###########################################
// Initialises the excess fow and the active block map
__global__ void InitialisationKernel(const float* edge_weights, const float* source_weights, const float* sink_weights, int* edge_capacities, int* excess_flow, int* active_block_map) 
{    
    int block_id = blockIdx.x;
    int thread_id = threadIdx.x;

    int block_y = block_id / c_block_count_x;
    int block_x = block_id - block_y * c_block_count_x;

    int thread_y = thread_id / BLOCK_SIZE;
    int thread_x = thread_id - thread_y * BLOCK_SIZE;

    int x = thread_x + block_x * BLOCK_SIZE;
    int y = thread_y + block_y * BLOCK_SIZE;

    if (x >= c_width) return;
    if (y >= c_height) return;

    int home = x + y * c_width;

    // writing edge capacities
    int edges = 0;

    #pragma unroll
    for (int i = 0; i < 4; i++)
    {
        int capacity = edge_weights[home + i * c_element_count] * HEIGHT_MAX;
        edges = set_edge(edges, i, capacity);
    }

    edge_capacities[home] = edges;

    // initialising excess flow
    excess_flow[home] = (sink_weights[home] - source_weights[home]) * HEIGHT_MAX;

    // initialising active block map
    if(thread_id == 0)
    {
        active_block_map[block_id] = block_id;
    }
}

//###########################################
// Disperses excess flow through the network based on edge capacities.
__global__ void PushKernel(const int* active_block_map, const int* height, int* edge_capacities, int* excess_flow) 
{    
    int block_id = blockIdx.x;
    int thread_id = threadIdx.x;
    
    int active_block = active_block_map[block_id];

    int block_y = active_block / c_block_count_x;
    int block_x = active_block - block_y * c_block_count_x;

    int thread_y = thread_id / BLOCK_SIZE;
    int thread_x = thread_id - thread_y * BLOCK_SIZE;

    int x = thread_x + block_x * BLOCK_SIZE;
    int y = thread_y + block_y * BLOCK_SIZE;

    if (x >= c_width) return;
    if (y >= c_height) return;

    int home = x + y * c_width;
    int home_height = height[home];
    int home_flow = excess_flow[home];
    int home_edges = edge_capacities[home];

    if (home_height > HEIGHT_MAX || home_flow <= 0) return;
    
    bool is_edge[4] = {y == c_height, x == c_width, y == 0, x == 0};
    int offsets[4] = {c_width, 1, -c_width, -1};

    #pragma unroll
    for (int i = 0; i < 4; i++) 
    {
        int neighbour = home + offsets[i];

        if (!is_edge[i] && height[neighbour] == home_height - 1) 
        {
            int neighbour_edges = edge_capacities[neighbour];

            int home_edge = get_edge(home_edges, i);
            int neighbour_edge = get_edge(neighbour_edges, i);

            float edge_flow = min(home_edge, home_flow);

            home_edges = set_edge(home_edges, i, home_edge - edge_flow);
            neighbour_edges = set_edge(neighbour_edges, i, neighbour_edge + edge_flow);

            excess_flow[home] -= edge_flow;
            excess_flow[neighbour] += edge_flow;

            edge_capacities[neighbour] = neighbour_edges;
        }
    }

    edge_capacities[home] = home_edges;
}

//###########################################
// Sets the height of each node based on excess flow 
__global__ void RelabelKernel(const int* active_block_map, const int* height_read, const int* edge_capacities, const int* excess_flow, int* height_write) 
{
    int block_id = blockIdx.x;
    int thread_id = threadIdx.x;
    
    int active_block = active_block_map[block_id];

    int block_y = active_block / c_block_count_x;
    int block_x = active_block - block_y * c_block_count_x;

    int thread_y = thread_id / BLOCK_SIZE;
    int thread_x = thread_id - thread_y * BLOCK_SIZE;

    int x = thread_x + block_x * BLOCK_SIZE;
    int y = thread_y + block_y * BLOCK_SIZE;

    if (x >= c_width) return;
    if (y >= c_height) return;

    int home = x + y * c_width;
    int home_height = height_read[home];
    int home_flow = excess_flow[home];
    int home_edges = edge_capacities[home];

    if (home_height > HEIGHT_MAX || home_flow <= 0) return;

    bool is_edge[4] = {y == c_height, x == c_width, y == 0, x == 0};
    int offsets[4] = {c_width, 1, -c_width, -1};

    int min_height = HEIGHT_MAX;

    #pragma unroll
    for (int i = 0; i < 4; i++) 
    {
        if (!is_edge[i] && get_edge(home_edges, i) > 0)
        {
            int neighbour = home + offsets[i];
            min_height = min(min_height, height_read[neighbour]+1);
        }
    }

    height_write[home] = min_height;
}

//###########################################
// Checks for any active nodes within block
__global__ void ActiveBlockCheckKernel(const int* height, const int* excess_flow, int* active_block_map) 
{
    int block_id = blockIdx.x;
    int thread_id = threadIdx.x;
    
    int block_y = block_id / c_block_count_x;
    int block_x = block_id - block_y * c_block_count_x;

    int thread_y = thread_id / BLOCK_SIZE;
    int thread_x = thread_id - thread_y * BLOCK_SIZE;

    int x = thread_x + block_x * BLOCK_SIZE;
    int y = thread_y + block_y * BLOCK_SIZE;

    if (x >= c_width) return;
    if (y >= c_height) return;

    int home = x + y * c_width;
    int home_height = height[home];
    int home_flow = excess_flow[home];

    bool active = home_height <= HEIGHT_MAX && home_flow > 0;
    bool any_active = __syncthreads_or(active);

    if(threadIdx.x == 0)
    {
        active_block_map[block_id] = any_active ? block_id : -1;
    } 
}

//###########################################

__global__ void TEMP_OUTPUT_DUMP(const int* height, const int* excess_flow, const int* edge_capacities, int* output) 
{
    int block_id = blockIdx.x;
    int thread_id = threadIdx.x;
    
    int block_y = block_id / c_block_count_x;
    int block_x = block_id - block_y * c_block_count_x;

    int thread_y = thread_id / BLOCK_SIZE;
    int thread_x = thread_id - thread_y * BLOCK_SIZE;

    int x = thread_x + block_x * BLOCK_SIZE;
    int y = thread_y + block_y * BLOCK_SIZE;

    if (x >= c_width) return;
    if (y >= c_height) return;

    int home = x + y * c_width;
    int home_height = height[home];
    int home_flow = excess_flow[home];
    int home_edges = edge_capacities[home];

    bool active = home_height <= HEIGHT_MAX && home_flow > 0;
    bool any_active = __syncthreads_or(active);

    output[home + 0 * c_element_count] = home_flow;
    output[home + 1 * c_element_count] = home_height;
    output[home + 2 * c_element_count] = get_edge(home_edges, 0);
    output[home + 3 * c_element_count] = get_edge(home_edges, 1);
    output[home + 4 * c_element_count] = get_edge(home_edges, 2);
    output[home + 5 * c_element_count] = get_edge(home_edges, 3);
    output[home + 6 * c_element_count] = any_active ? 1 : 0;
}

//###########################################

torch::Tensor GraphCut_Cuda(torch::Tensor edge_weights_tensor, torch::Tensor source_weights_tensor, torch::Tensor sink_weights_tensor)
{
    TensorDescription desc = TensorDescription(edge_weights_tensor);

    float *edge_weights = edge_weights_tensor.data_ptr<float>();
    float *source_weights = source_weights_tensor.data_ptr<float>();
    float *sink_weights = sink_weights_tensor.data_ptr<float>();

    int dimensions = desc.dimensions;
    int width = desc.sizes[0];
    int height = desc.sizes[1];
    int element_count = width * height;

    int block_count_x = int(width / BLOCK_SIZE) + 1;
    int block_count_y = int(height / BLOCK_SIZE) + 1;
    int block_count = block_count_x * block_count_y;

    int block_thread_count = pow(BLOCK_SIZE, dimensions);

    int *active_block_map_host, *active_block_map;
    active_block_map_host = new int[block_count];
    hipMalloc(&active_block_map, block_count * sizeof(int));

    int *edge_capacities;
    hipMalloc(&edge_capacities, element_count * sizeof(int));

    int *excess_flow;
    hipMalloc(&excess_flow, element_count * sizeof(int));

    int *height_read, *height_write, *height_swap;
    hipMalloc(&height_read, element_count * sizeof(int));
    hipMalloc(&height_write, element_count * sizeof(int));
    
    hipMemcpyToSymbol(HIP_SYMBOL(c_width), &width, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(c_height), &height, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(c_block_count_x), &block_count_x, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(c_element_count), &element_count, sizeof(int));

    int active_block_count = block_count;

    // Initialisation...
    InitialisationKernel<<<active_block_count, block_thread_count>>>(edge_weights, source_weights, sink_weights, edge_capacities, excess_flow, active_block_map);

    int stuck_counter = 0;

    // Iterating while any blocks are active...
    while (active_block_count > 0 && stuck_counter < 50)
    {
        py::print(active_block_count);

        PushKernel<<<active_block_count, block_thread_count>>>(active_block_map, height_read, edge_capacities, excess_flow);

        RelabelKernel<<<active_block_count, block_thread_count>>>(active_block_map, height_read, edge_capacities, excess_flow, height_write);
        height_swap = height_read; 
        height_read = height_write; 
        height_write = height_swap;

        ActiveBlockCheckKernel<<<block_count, block_thread_count>>>(height_read, excess_flow, active_block_map);

        //######################################
        // host side stream compaction for now

        hipMemcpy(active_block_map_host, active_block_map, block_count * sizeof(int), hipMemcpyDeviceToHost);

        int previous_active_block_count = active_block_count;
        active_block_count = 0;

        for (int i = 0; i < block_count; i++)
        {
            if(active_block_map_host[i] != -1) 
            {
                active_block_map_host[active_block_count] = active_block_map_host[i];
                active_block_count++;
            }
        }

        hipMemcpy(active_block_map, active_block_map_host, active_block_count * sizeof(int), hipMemcpyHostToDevice);
        
        //######################################

        if (active_block_count == previous_active_block_count)
        {
            stuck_counter++;
        }
        else
        {
            stuck_counter = 0;
        }
    }

    torch::Tensor output_tensor = torch::zeros({1, 7, width, height}, torch::dtype(torch::kInt32).device(torch::kCUDA, 0));

    TEMP_OUTPUT_DUMP<<<block_count, block_thread_count>>>(height_read, excess_flow, edge_capacities, output_tensor.data_ptr<int>());

    return output_tensor;
}
