#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "push_relabel.cuh"

torch::Tensor GraphCut_Cuda(torch::Tensor input_graph, torch::Tensor source_weights, torch::Tensor sink_weights, int iterations, int a, int b)
{
    PushRelabel push_relabel = PushRelabel(input_graph, source_weights, sink_weights);

    torch::Tensor output = push_relabel.Execute(iterations, a, b);

    return output;
}
