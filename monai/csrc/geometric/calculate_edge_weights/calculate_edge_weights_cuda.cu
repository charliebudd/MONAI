#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include "utils/tensor_description.h"

#define BLOCK_SIZE 32

__constant__ int cWidth;
__constant__ int cHeight;
__constant__ int cChannelStride;

__global__ void CalculateEdgeWeightsKernel(const float* input, float* output) {
    int x = threadIdx.x + blockIdx.x * BLOCK_SIZE;
    int y = threadIdx.y + blockIdx.y * BLOCK_SIZE;

    if (x >= cWidth) return;
    if (y >= cHeight) return;

    int home = x + y * cWidth;
    float home_value = input[home];

    bool edge[4] = {y == cHeight, x == cWidth, y == 0, x == 0};
    int offsets[4] = {cWidth, 1, -cWidth, -1};

    #pragma unroll
    for (int i = 0; i < 4; i++){
        float diff = edge[i] ? 0 :  home_value - input[home + offsets[i]];
        output[home + i * cChannelStride] = exp(-(diff * diff));
    }
}

torch::Tensor CalculateEdgeWeights_Cuda(torch::Tensor input_tensor)
{
    TensorDescription desc = TensorDescription(input_tensor);

    torch::Tensor output_tensor = torch::zeros({desc.batchCount, 4, desc.sizes[0], desc.sizes[1]}, input_tensor.device());

    hipMemcpyToSymbol(HIP_SYMBOL(cWidth), &desc.sizes[0], sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(cHeight), &desc.sizes[1], sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(cChannelStride), &desc.channelStride, sizeof(int));

    int block_count_x = int(desc.sizes[0] / BLOCK_SIZE) + 1;
    int block_count_y = int(desc.sizes[1] / BLOCK_SIZE) + 1;
    int block_count_z = desc.batchCount;

    dim3 block_count = dim3(block_count_x, block_count_y, block_count_z);
    dim3 block_size = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

    CalculateEdgeWeightsKernel<<<block_count, block_size>>>(input_tensor.data_ptr<float>(), output_tensor.data_ptr<float>());

    return output_tensor;
}
